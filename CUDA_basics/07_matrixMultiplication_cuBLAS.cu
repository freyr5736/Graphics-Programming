#include "hip/hip_runtime.h"
// Matrix multiplication using cuBLAS with unified memory and prefetching

#include <stdio.h>
#include <cassert>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using std::cout;

// Matrix dimensions (change as needed)
#define N 512  // Rows of A and C
#define M 512  // Columns of A and Rows of B
#define K 512  // Columns of B and C

// Main function
int main() {
    size_t bytes = N * M * sizeof(float);  // Size for A
    size_t bytesB = M * K * sizeof(float); // Size for B
    size_t bytesC = N * K * sizeof(float); // Size for C

    // Declare unified memory pointers
    float *A, *B, *C;

    // Allocate unified memory
    hipMallocManaged(&A, bytes);
    hipMallocManaged(&B, bytesB);
    hipMallocManaged(&C, bytesC);

    // Get device ID for prefetching
    int id;
    hipGetDevice(&id);

    // Set memory hints and prefetch
    hipMemAdvise(A, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(B, bytesB, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(C, bytesC, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);

    // Initialize matrices with random values
    for (int i = 0; i < N * M; i++) A[i] = static_cast<float>(rand() % 100);
    for (int i = 0; i < M * K; i++) B[i] = static_cast<float>(rand() % 100);

    // Prefetch to GPU
    hipMemAdvise(A, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(B, bytesB, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(A, bytes, id);
    hipMemPrefetchAsync(B, bytesB, id);
    hipMemPrefetchAsync(C, bytesC, id);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Scalar values for cuBLAS (alpha, beta)
    const float alpha = 1.0f, beta = 0.0f;

    // Perform matrix multiplication using cuBLAS: C = A * B
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                   K, N, M,  // Output matrix (K x N), Input matrices (M x K) and (N x M)
                   &alpha, 
                   B, K,  // B (M x K) stored in column-major
                   A, M,  // A (N x M) stored in column-major
                   &beta, 
                   C, K);  // C (N x K) stored in column-major

    // Synchronize to ensure computation is complete
    hipDeviceSynchronize();

    // Prefetch result back to CPU
    hipMemPrefetchAsync(C, bytesC, hipCpuDeviceId);

    // Verify results on CPU
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < K; j++) {
            float sum = 0.0f;
            for (int k = 0; k < M; k++) {
                sum += A[i * M + k] * B[k * K + j];
            }
            assert(fabs(C[i * K + j] - sum) < 1e-3);
        }
    }

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    // Free unified memory
    hipFree(A);
    hipFree(B);
    hipFree(C);

    cout << "EXECUTION SUCCESSFUL!\n";

    return 0;
}
