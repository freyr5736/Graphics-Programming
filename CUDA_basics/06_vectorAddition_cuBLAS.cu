#include "hip/hip_runtime.h"
// This program computes the sum of two N-element vectors using cuBLAS and unified memory prefetch

#include <stdio.h>
#include <cassert>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using std::cout;

// Main function
int main() {
    // Array size of 2^16 (65536 elements)
    const int N = 1 << 16;
    size_t bytes = N * sizeof(float);

    // Declare unified memory pointers
    float *a, *b, *c;

    // Allocate unified memory
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Get the device ID for prefetching calls
    int id;
    hipGetDevice(&id);

    // Set memory hints and prefetch
    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, id);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        a[i] = static_cast<float>(rand() % 100);
        b[i] = static_cast<float>(rand() % 100);
    }

    // Prefetch 'a' and 'b' arrays to the GPU
    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Scalar for SAXPY operation
    const float alpha = 1.0f;

    // Copy 'b' to 'c' before performing cuBLAS operation
    hipMemcpy(c, b, bytes, hipMemcpyDeviceToDevice);

    // Perform vector addition using cuBLAS: c = a + b (by modifying c)
    hipblasSaxpy(handle, N, &alpha, a, 1, c, 1);

    // Synchronize device to ensure computation is finished
    hipDeviceSynchronize();

    // Verify the result on the CPU
    for (int i = 0; i < N; i++) {
        assert(fabs(c[i] - (a[i] + b[i])) < 1e-5);
    }

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    cout << "EXECUTION SUCCESSFUL!\n";

    return 0;
}
